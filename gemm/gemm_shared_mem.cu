#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <cmath>
#include <hipblas.h>

#define DEBUG 0

/*
 compute C = alpha*A*B + beta*C
 MxN = MxK + KxN
*/

template<const uint BLOCKSIZE>
__global__ void sgemm(int M, int K, int N, float alpha, float beta, const float* A, const float* B, float* C)
{

    A += blockIdx.x * BLOCKSIZE * K;
    B += blockIdx.y * BLOCKSIZE;
    C += blockIdx.x * BLOCKSIZE * N + blockIdx.y * BLOCKSIZE;

    // one thread block compute thread block size of C
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    int x = threadIdx.x / BLOCKSIZE;
    int y = threadIdx.x % BLOCKSIZE;


    float c = 0;
    for(int block = 0; block < K/BLOCKSIZE; ++block)
    {
        As[x * BLOCKSIZE + y] = A[x * K + y];
        Bs[x * BLOCKSIZE + y] = B[x * N + y];

        // sync to make sure shared memory are fully loaded by all threads in a thread block
        __syncthreads();

        A += BLOCKSIZE;
        B += BLOCKSIZE * N;
        // inner block loop
        for(int k = 0; k < BLOCKSIZE; ++k)
        {
            c += As[x * BLOCKSIZE + k] * Bs[y + k * BLOCKSIZE];
        }

        // sync to make sure all threads in a thread block have completed the partial sum
        // shared memory are all consumed before the next load
        __syncthreads();
    }

    C[x * N + y] = alpha*c + beta*C[x * N + y];
}

int main()
{
for(int i = 1024; i <= 2048; i += 1024)
{
    int M = i;
    int N = i;
    int K = i;
    float alpha = 1;
    float beta = 0;
  
    auto a_host = new float[M*K];
    auto b_host = new float[K*N];
    auto c_host = new float[M*N];
    auto c_ref_host = new float[M*N];
    std::cout << "M=" << M << ", N=" << N << ", K=" <<K << std::endl;
    srand((unsigned)time(NULL));
#if DEBUG
    std::cout << "a_host" << std::endl;
#endif
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < K; ++j)
        {
            a_host[i*M + j] = static_cast<float>(rand() % 100);
          #if DEBUG
            std::cout << a_host[i*M + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }

#if DEBUG
    std::cout << "b_host" << std::endl;
#endif
    for(int i = 0; i < K; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            b_host[i*K + j] = static_cast<float>(rand() % 100);
          #if DEBUG
            std::cout << b_host[i*K + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }
#if DEBUG
    std::cout << "c_ref" << std::endl;
#endif
#if DEBUG
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            int c = 0;
            for(int k = 0; k < K; ++k)
            {
                c += a_host[i*M + k] * b_host[k*K + j];
            }
            c_ref_host[i*M + j] = alpha*c + beta*c_ref_host[i*M + j];
          #if DEBUG
            std::cout << c_ref_host[i*M + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }
#endif

    float* a_device;
    float* b_device;
    float* c_device;
    float* c_ref_device;
    hipMalloc(&a_device, sizeof(float)*M*K);
    hipMalloc(&b_device, sizeof(float)*K*N);
    hipMalloc(&c_device, sizeof(float)*M*N);
    hipMalloc(&c_ref_device, sizeof(float)*M*N);

    hipMemcpy(a_device, a_host, sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, sizeof(float)*K*N, hipMemcpyHostToDevice);

/*
cublasStatus_t cublasSgemm(cublasHandle_t handle,
               cublasOperation_t transa, cublasOperation_t transb,
			   int m, int n, int k,
			   const float *alpha,
			   const float *A, int lda,
		       const float *B, int ldb,										       
               const float *beta,
		       float *C, int ldc)
*/



    // create handle
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
	    std::cout << "CUBLAS initialization failure" << std::endl;
    }
    stat = hipblasSgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
			   N, M, K,
			   &alpha,
			   b_device, N,
		           a_device, K,
			   &beta,
			   c_ref_device, N);

    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
	    std::cout << "CUBLAS Sgemm execution failure" << std::endl;
    }

    //
    dim3 blocksPerGrid((M+31)/32, (N+31)/32);
    // use 1024 threads per block
    // one thread compute one output element of C
    dim3 threadsPerBlock(32*32);
    // warm up
    sgemm<32><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);

    float elapsed_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    int repeat_times = 10;
    hipEventRecord(start);
    for(int i = 0; i < repeat_times; ++i)
    {
        sgemm<32><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);
    }

    hipError_t err;
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "kernel launch or execute failure" << std::endl;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    hipEventRecord(end);
    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    elapsed_time /= 1000.0; // seconds

    std::cout << "Average elapsed time: " << elapsed_time/repeat_times << " second(s), performance: "
	    << (1.0e-9)*2*M*K*N*repeat_times/elapsed_time << " GFLOPS. Memory bandwith: "
        << (1.0e-9)*4*(M*K + K*N + M*N)*repeat_times/elapsed_time << " GB/s" << std::endl;

    hipMemcpy(c_host, c_device, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    float elapsed_time_cublas;
    hipEvent_t start_cublas, end_cublas;
    hipEventCreate(&start_cublas);
    hipEventCreate(&end_cublas);

    hipEventRecord(start_cublas);
    for(int i = 0; i < repeat_times; ++i)
    {
          hipblasSgemm(handle,
              HIPBLAS_OP_N, HIPBLAS_OP_N,
			   N, M, K,
			   &alpha,
			   b_device, N,
		           a_device, K,
			   &beta,
			   c_ref_device, N);
    }
    hipEventRecord(end_cublas);
    hipEventSynchronize(start_cublas);
    hipEventSynchronize(end_cublas);
    hipEventElapsedTime(&elapsed_time_cublas, start_cublas, end_cublas);
    elapsed_time_cublas /= 1000.0; // seconds
    std::cout << "Cublas Average elapsed time: " << elapsed_time_cublas << " second(s), performance: "
	    << (1.0e-9)*2*M*K*N*repeat_times/elapsed_time_cublas << " GFLOPS. Memory bandwidth: "
        << (1.0e-9)*4*(M*K + K*N + M*N)*repeat_times/elapsed_time_cublas << " GB/s" << std::endl;

    hipMemcpy(c_ref_host, c_ref_device, sizeof(float)*M*N, hipMemcpyDeviceToHost);
 

    // verify
    bool flag = true;
#if 1
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            if(abs(c_ref_host[i*M + j]-c_host[i*M + j]>0.01))
            {
                std::cout << "(" << i << "," << j << ") " <<"diff(cublas, custom) " << c_ref_host[i*M + j]-c_host[i*M + j] << std::endl;
                flag = false;
            }
        }
    }
#endif


    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
    hipFree(c_ref_device);

    delete[] a_host;
    delete[] b_host;
    delete[] c_host;
    delete[] c_ref_host;

    if(flag)
    {
        std::cout << "compute pass" << std::endl;
    }else
    {
        std::cout << "compute fail" << std::endl;
    }
}

}
