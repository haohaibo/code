#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel() {
    extern __shared__ int sharedMem[]; // 动态分配共享内存
}

int main() {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(kernel));

    std::cout << "Shared Memory Size (Static): " << attr.sharedSizeBytes << " bytes" << std::endl;
    return 0;
}