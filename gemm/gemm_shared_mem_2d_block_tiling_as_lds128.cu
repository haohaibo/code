#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <cmath>
#include <hipblas.h>
#include <stdio.h>

#define DEBUG 0

// default set to 10
#define REPEAT_TIMES 10

// default set to 1
#define WARM_UP 1

/*
 compute C = alpha*A*B + beta*C
 MxN = MxK + KxN
*/

template<const uint BM, const uint BK, const uint BN, const uint TM, const uint TN>
__global__ void sgemm(int M, int K, int N, float alpha, float beta, const float* A, const float* B, float* C)
{
    A += blockIdx.x * BM * K;
    B += blockIdx.y * BN;
    C += blockIdx.x * BM * N + blockIdx.y * BN;

    // one thread block compute thread block size of C
    __shared__ float As[BM * BK];
    __shared__ float Bs[BK * BN];

    //int xa = threadIdx.x / (BK/TN);
    //int ya = threadIdx.x % (BK/TN);

    int elementsLoadPerThread = (BK * TM * TN) / BM;

    int xb = threadIdx.x / (BN/TN);
    int yb = threadIdx.x % (BN/TN);


    int colA = threadIdx.x % BK;
    int rowA = threadIdx.x / BK;

    int colB = threadIdx.x % BN;
    int rowB = threadIdx.x / BN;

    float c[TM*TN] = {0.0};

    float regAs[TM] = {0.0};
    float regBs[TN] = {0.0};
    for(int block = 0; block < K/BK; ++block)
    {
    #if 0
        for(int t = 0; t < TM; ++t)
        {
            //As[xa * BK + ya * TN + t] = A[xa * K + yb * TN + t];

            As[(xb * TM + t) * BK + yb] = A[(xb * TM + t) * K + yb];
        }
    #endif
        for(int t = 0; t < elementsLoadPerThread; ++t)
        {
            //As[((rowA * elementsLoadPerThread + t) * BK) + colA] = A[((rowA * elementsLoadPerThread + t) * K) + colA];

            As[(colA * BM) + (rowA * elementsLoadPerThread + t)] = A[((rowA * elementsLoadPerThread + t) * K) + colA];
        }

        for(int t = 0; t < elementsLoadPerThread; ++t)
        {
            //Bs[xb * BN + yb * TN + t] = B[xb * N + yb * TN + t];

            //Bs[(t * BN) + threadIdx.x] = B[(t * N) + threadIdx.x];

            Bs[((rowB * elementsLoadPerThread + t) * BN) + colB] = B[((rowB * elementsLoadPerThread + t) * N) + colB];
        }

        // sync to make sure shared memory are fully loaded by all threads in a thread block
        __syncthreads();

        A += BK;
        B += BK * N;

        // inner block loop
        for(int k = 0; k < BK; ++k)
        {
            for(int i = 0; i < TM; ++i)
            {
                //regAs[i] = As[(xb * TM + i) * BK + k];

                regAs[i] = As[(xb * TM + i) + k * BM];
            }

            for(int j = 0; j < TN; ++j)
            {
                regBs[j] = Bs[(yb * TN + j) + k * BN];
            }

            for(int tm = 0; tm < TM; ++tm)
            {
                for(int tn = 0; tn < TN; ++tn)
                {
                    //c[tm * TN + tn] += As[(xb * TM + tm) * BK + k] * Bs[(yb * TN + tn) + k * BN];
                    c[tm * TN + tn] += regAs[tm] * regBs[tn];
                }
            }
        }
        // sync to make sure all threads in a thread block have completed the partial sum
        // shared memory are all consumed before the next load
        __syncthreads();
    }

    for(int tm = 0; tm < TM; ++tm)
    {
        for(int tn = 0; tn < TN; ++tn)
        {
            C[(xb * TM + tm) * N + yb * TN + tn] = alpha*c[tm * TN + tn] + beta*C[(xb * TM + tm) * N + yb * TN + tn];
        }
    }

}

int main()
{
for(int i = 1024; i <= 4096; i += 1024)
{
    int M = i;
    int N = i;
    int K = i;
    float alpha = 1;
    float beta = 0;
  
    auto a_host = new float[M*K];
    auto b_host = new float[K*N];
    auto c_host = new float[M*N];
    auto c_ref_host = new float[M*N];
    std::cout << "M=" << M << ", N=" << N << ", K=" <<K << std::endl;
    srand((unsigned)time(NULL));
#if DEBUG
    std::cout << "a_host" << std::endl;
#endif
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < K; ++j)
        {
            a_host[i*M + j] = static_cast<float>(rand() % 5);
          #if DEBUG
            std::cout << a_host[i*M + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }

#if DEBUG
    std::cout << "b_host" << std::endl;
#endif
    for(int i = 0; i < K; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            b_host[i*K + j] = static_cast<float>(rand() % 5);
          #if DEBUG
            std::cout << b_host[i*K + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }
#if DEBUG
    std::cout << "c_ref_host" << std::endl;
#endif
#if DEBUG
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            int c = 0;
            for(int k = 0; k < K; ++k)
            {
                c += a_host[i*M + k] * b_host[k*K + j];
            }
            c_ref_host[i*M + j] = alpha*c + beta*c_ref_host[i*M + j];
          #if DEBUG
            std::cout << c_ref_host[i*M + j] << " ";
          #endif
        }
        #if DEBUG
        std::cout << std::endl;
        #endif
    }
#endif

    float* a_device;
    float* b_device;
    float* c_device;
    float* c_ref_device;
    hipMalloc(&a_device, sizeof(float)*M*K);
    hipMalloc(&b_device, sizeof(float)*K*N);
    hipMalloc(&c_device, sizeof(float)*M*N);
    hipMalloc(&c_ref_device, sizeof(float)*M*N);

    hipMemcpy(a_device, a_host, sizeof(float)*M*K, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, sizeof(float)*K*N, hipMemcpyHostToDevice);

/*
cublasStatus_t cublasSgemm(cublasHandle_t handle,
               cublasOperation_t transa, cublasOperation_t transb,
			   int m, int n, int k,
			   const float *alpha,
			   const float *A, int lda,
		       const float *B, int ldb,										       
               const float *beta,
		       float *C, int ldc)
*/



    // create handle
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
	    std::cout << "CUBLAS initialization failure" << std::endl;
    }
#if WARM_UP
    stat = hipblasSgemm(handle,
                           HIPBLAS_OP_N, HIPBLAS_OP_N,
			   N, M, K,
			   &alpha,
			   b_device, N,
		           a_device, K,
			   &beta,
			   c_ref_device, N);

    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
	    std::cout << "CUBLAS Sgemm execution failure" << std::endl;
    }
#endif

    // 1 thread compute 8x8 output elements acrossing M dimension
    const uint TM = 8, TN = 8;
    const uint BK = 8;

    float elapsed_time;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    if(M >= 128 && N >= 128)
    {
    const uint BM = 128, BN = 128;

    //
    dim3 blocksPerGrid((M+BM-1)/BM, (N+BN-1)/BN);
    // one thread compute 8x8 output element of C
    dim3 threadsPerBlock((BM*BN)/(TM*TN));

#if WARM_UP
    // warm up
    sgemm<BM, BK, BN, TM, TN><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);
#endif

    hipEventRecord(start);
    for(int i = 0; i < REPEAT_TIMES; ++i)
    {
        sgemm<BM, BK, BN, TM, TN><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);
    }
    hipEventRecord(end);

    }else
    {
    const uint BM = 64, BN = 64;
    //
    dim3 blocksPerGrid((M+BM-1)/BM, (N+BN-1)/BN);
    // one thread compute 8x8 output element of C
    dim3 threadsPerBlock((BM*BN)/(TM*TN));

#if WARM_UP
    // warm up
    sgemm<BM, BK, BN, TM, TN><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);
#endif

    hipEventRecord(start);
    for(int i = 0; i < REPEAT_TIMES; ++i)
    {
        sgemm<BM, BK, BN, TM, TN><<<blocksPerGrid, threadsPerBlock>>>(M, K, N, alpha, beta, a_device, b_device, c_device);
    }
    hipEventRecord(end);
    }


    hipError_t err;
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        std::cout << "kernel launch or execute failure" << std::endl;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    hipEventSynchronize(start);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, start, end);
    elapsed_time /= 1000.0; // seconds

    std::cout << "Average elapsed time: " << elapsed_time/REPEAT_TIMES << " second(s), performance: "
	    << (1.0e-9)*2*M*K*N*REPEAT_TIMES/elapsed_time << " GFLOPS. Memory bandwith: "
        << (1.0e-9)*4*(M*K + K*N + M*N)*REPEAT_TIMES/elapsed_time << " GB/s" << std::endl;

    hipMemcpy(c_host, c_device, sizeof(float)*M*N, hipMemcpyDeviceToHost);

    float elapsed_time_cublas;
    hipEvent_t start_cublas, end_cublas;
    hipEventCreate(&start_cublas);
    hipEventCreate(&end_cublas);

    hipEventRecord(start_cublas);
    for(int i = 0; i < REPEAT_TIMES; ++i)
    {
    stat = hipblasSgemm(handle,
               HIPBLAS_OP_N, HIPBLAS_OP_N,
			   N, M, K,
			   &alpha,
			   b_device, N,
		           a_device, K,
			   &beta,
			   c_ref_device, N);
    }

    if(stat != HIPBLAS_STATUS_SUCCESS)
    {
	    std::cout << "CUBLAS Sgemm execution failure" << std::endl;
    }
    hipEventRecord(end_cublas);
    hipEventSynchronize(start_cublas);
    hipEventSynchronize(end_cublas);
    hipEventElapsedTime(&elapsed_time_cublas, start_cublas, end_cublas);
    elapsed_time_cublas /= 1000.0; // seconds
    std::cout << "Cublas Average elapsed time: " << elapsed_time_cublas << " second(s), performance: "
	    << (1.0e-9)*2*M*K*N*REPEAT_TIMES/elapsed_time_cublas << " GFLOPS. Memory bandwidth: "
        << (1.0e-9)*4*(M*K + K*N + M*N)*REPEAT_TIMES/elapsed_time_cublas << " GB/s" << std::endl;

    hipMemcpy(c_ref_host, c_ref_device, sizeof(float)*M*N, hipMemcpyDeviceToHost);
 

#if DEBUG
    std::cout << "c_host" << std::endl;
#endif
#if DEBUG
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            std::cout << c_host[i*M + j] << " ";
        }
        std::cout << std::endl;
    }
#endif


    // verify
    bool flag = true;
#if 1
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            if(abs(c_ref_host[i*M + j]-c_host[i*M + j]>0.01))
            {
                std::cout << "(" << i << "," << j << ") " <<"diff(cublas, custom) " << c_ref_host[i*M + j]-c_host[i*M + j] << std::endl;
                flag = false;
            }
        }
    }
#endif


    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
    hipFree(c_ref_device);

    delete[] a_host;
    delete[] b_host;
    delete[] c_host;
    delete[] c_ref_host;

    if(flag)
    {
        std::cout << "compute pass" << std::endl;
    }else
    {
        std::cout << "compute fail" << std::endl;
    }
}

}
