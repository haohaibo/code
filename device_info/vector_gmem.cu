#include <iostream>
#include <hip/hip_runtime.h>

#define N 1024

__global__ void deviceCopy(int* dOut, int* dIn, int Count)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(int i = tid; i < Count/4; i += blockDim.x * gridDim.x)
    {
        reinterpret_cast<int4*>(dOut)[i] = reinterpret_cast<int4*>(dIn)[i];
    }
}

int main()
{
    int* hIn = new int[N];
    int* hOut = new int[N];
    
    for(int i = 0; i < N; ++i)
    {
        hIn[i] = i;
    }

    int* dIn;
    hipMalloc(&dIn, N * sizeof(int));
    int* dOut;
    hipMalloc(&dOut, N * sizeof(int));

    hipMemcpy(dIn, hIn, N * sizeof(int), hipMemcpyHostToDevice);

    int threads = 128;

    dim3 blocksPerGrid((N/4 + threads -1) / threads);
    dim3 threadsPerBlock(threads);

    deviceCopy<<<blocksPerGrid, threadsPerBlock>>>(dOut, dIn, N);
    hipMemcpy(hOut, dOut, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; ++i)
    {
        if((hOut[i] - hIn[i]) > 0.01)
        {
            std::cout << "output mismatch!" << std::endl;
            break;
        }
    }
}
